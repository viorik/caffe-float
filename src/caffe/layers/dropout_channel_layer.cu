#include "hip/hip_runtime.h"
#include <algorithm>
#include <limits>
#include <vector>

#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/syncedmem.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {


template <typename Dtype>
__global__ void DropoutChannelForward(const int n, const Dtype* in,
    const Dtype* mask, const Dtype threshold, const float scale,
    Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] * (mask[index] > threshold) * scale;
  }
}

template <typename Dtype>
void DropoutChannelLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  if (Caffe::phase() == Caffe::TRAIN) {
    const Dtype* const_vec = rand_vec_.gpu_data();
    Dtype* vec = rand_vec_.mutable_gpu_data();
    Dtype* mask = rand_mat_.mutable_gpu_data();

    caffe_gpu_rng_uniform(rand_vec_.count(), Dtype(0), Dtype(1), vec);

    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, 
                          N_ * C_, H_ * W_, 1, Dtype(1),
                          const_vec, spatial_sum_multiplier_.gpu_data(),
                          Dtype(0), mask);

    // set thresholds
    // NOLINT_NEXT_LINE(whitespace/operators)
    DropoutChannelForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, mask, threshold_, scale_, top_data);
    CUDA_POST_KERNEL_CHECK;
  } else {
    caffe_copy(count, bottom_data, top_data);
  }
}

template <typename Dtype>
__global__ void DropoutChannelBackward(const int n, const Dtype* in_diff,
    const Dtype* mask, const Dtype threshold, const float scale,
    Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * scale * (mask[index] > threshold);
  }
}

template <typename Dtype>
void DropoutChannelLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    if (Caffe::phase() == Caffe::TRAIN) {
      const Dtype* mask = rand_mat_.gpu_data();
      const int count = bottom[0]->count();
      // NOLINT_NEXT_LINE(whitespace/operators)
      DropoutChannelBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(
          count, top_diff, mask, threshold_, scale_, bottom_diff);
      CUDA_POST_KERNEL_CHECK;
    } else {
      caffe_copy(top[0]->count(), top_diff, bottom_diff);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(DropoutChannelLayer);


}  // namespace caffe
